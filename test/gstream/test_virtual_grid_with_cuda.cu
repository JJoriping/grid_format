#include "hip/hip_runtime.h"
#include "../grid_format_test.h"
#include <gstream/test/gstream_test_defines.h>
#include <gstream/framework/execution_model/virtual_grid_service.h>
#include <gstream/grid_dataset.h>
#include <mixx/string.h>
#include <hip/hip_runtime_api.h>

using grid_format = as20000102_grid;
using virtual_gb = gstream::virtual_grid_block<grid_format, 4>;

static __device__ int device_edge_counter[1];

static __global__ void virtual_grid_traversal(virtual_gb* vgb) {
    unsigned current = 0;
    for (auto vgb_iter = vgb->begin(); vgb_iter != vgb->end(); ++vgb_iter) {
        grid_format::sparse_gb* gb = *vgb_iter;
        for (auto gb_iter = gb->begin(); gb_iter != gb->end(); ++gb_iter) {
            grid_format::gb_column column = *gb_iter;
            //gstream::encoded_edge encoded;
            //encoded.dst = column.colidx;
            for (auto col_iter = column.begin(); col_iter != column.end(); ++col_iter) {
                //encoded.src = *col_iter;
                device_edge_counter[0] += 1;
            }
        }
    }
}

TEST_CASE("virtual grid with cuda", GSTREAM_TEST_TAG_CUDA GSTREAM_TEST_TAG_GRID_FORMAT) {
    char input_dir[FILENAME_MAX];
    char current_dir[FILENAME_MAX];
    mixx::dirname_unix(current_dir, FILENAME_MAX, __FILE__);
    snprintf(input_dir, FILENAME_MAX, "%s/../%s", current_dir, "test_data");

    gstream::in_memory_grid_dataset dataset_host;
    REQUIRE(dataset_host.import("as20000102", input_dir) == gstream::GStreamSuccess);

    gstream::in_memory_device_grid_dataset dataset_dev;
    REQUIRE(dataset_dev.import(dataset_host) == gstream::GStreamSuccess);

    // init counter
    {
        int zero = 0;
        REQUIRE(hipMemcpyToSymbol(HIP_SYMBOL(device_edge_counter), &zero, sizeof(int), 0) == hipSuccess);
    }

    // allocate a deivce vgb buffer
    virtual_gb vgb;
    virtual_gb* vgb_dev;
    {
        REQUIRE(hipMalloc((void**)&vgb_dev, sizeof(virtual_gb)) == hipSuccess);
    }

    auto& info = dataset_host.info;
    for (uint32_t x = 0; x < info.dim.x; ++x) {
        for (uint32_t y = 0; y < info.dim.y; ++y) {
            gstream::gbi_node* gbi = dataset_dev.gbi(gstream::gbid_t{ x, y });
            if (dataset_dev.gbi(gstream::gbid_t{ x, y }) == nullptr)
                continue; // skip
            vgb.add_gbi(gbi);
            if (vgb.size() == 4) {
                hipMemcpy(vgb_dev, &vgb, sizeof(virtual_gb), hipMemcpyHostToDevice);
                virtual_grid_traversal<<<1,1>>>(vgb_dev);
                vgb.clear();
            }
        }
    }
    REQUIRE(hipDeviceSynchronize() == hipSuccess);

    // release the device vgb buffer
    REQUIRE(hipFree(vgb_dev) == hipSuccess);

    // test 
    {
        int result = 0;
        REQUIRE( hipMemcpyFromSymbol(&result, HIP_SYMBOL(device_edge_counter), sizeof(int), 0) == hipSuccess);
        REQUIRE( (uint64_t)result == dataset_host.info.edge_count );
        printf("result is %d\n", result);
    }
}